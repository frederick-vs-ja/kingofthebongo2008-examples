#include "hip/hip_runtime.h"
﻿#include "precompiled.h"

#include <cstdint>
#include <fstream>
#include <iostream>
#include <memory>

#include <util/util_memory.h>

#include <hip/hip_runtime.h>
#include <>
#include <device_functions_decls.h>
#include <hip/hip_vector_types.h>

#include <jxr/cuda_helper.h>
#include <jxr/jxr_transforms.h>
#include <jxr/jxr_analysis.h>

#include <jxr/jxr_filter.h>
#include <jxr/jxr_prefilter.h>
#include <jxr/jxr_overlapfilter.h>
#include <jxr/jxr_pct.h>
#include <jxr/jxr_decompose.h>

#include <os/windows/com_initializer.h>

#include "img_images.h"
#include "img_loader.h"

namespace example
{
    class cuda_initializer
    {
        public:
        cuda_initializer()
        {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cuda::throw_if_failed<cuda::exception> (  hipSetDevice(0) );
        }

        ~cuda_initializer()
        {
            // hipDeviceReset must be called before exiting in order for profiling and
            // tracing tools such as Nsight and Visual Profiler to show complete traces.
            cuda::throw_if_failed<cuda::exception> ( hipDeviceReset() );
        }
    };
}

namespace example
{
    struct rgb 
    {
        uint8_t color[3];
    };

    __global__ void scale_decompose_ycocg_kernel( const rgb* in, uint32_t* y_color, uint32_t* co_color, uint32_t* cg_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_co = element->color[1];
        jpegxr::transforms::pixel b_cg = element->color[2];

        using namespace jpegxr::transforms;

        scale_bias_bd8_analysis< no_scale, bd8 >(&r_y, &g_co, &b_cg);
        rgb_2_ycocg(&r_y, &g_co, &b_cg );

        y_color [ row * write_pitch + col ] = r_y;
        co_color[ row * write_pitch + col ] = g_co;
        cg_color[ row * write_pitch + col ] = b_cg;
    }

    std::shared_ptr< ycocg_image > make_ycocg ( std::shared_ptr<image> image ) 
    {
        auto w         = image->get_width();
        auto h         = image->get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = cuda::make_memory_buffer ( size );
        auto co_buffer = cuda::make_memory_buffer ( size );
        auto cg_buffer = cuda::make_memory_buffer ( size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_decompose_ycocg_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<rgb*> ( image->get() ), *y_buffer, *co_buffer, *cg_buffer, image->get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );


        /*
        //debug purposes
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), co_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), cg_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        */

        return std::make_shared<ycocg_image> ( make_image_2d ( y_buffer, w, h, w ), make_image_2d (co_buffer, w, h, w) , make_image_2d( cg_buffer, w, h, w ) ) ;
    }

    __global__ void scale_compose_ycocg_kernel( const uint32_t* y_color, const uint32_t* u_color, const uint32_t* v_color, rgb* out, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<rgb*> (  (uint8_t*) out + ( row * write_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y = y_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel g_u = u_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel b_v = v_color[ row * read_pitch + col ];

        using namespace jpegxr::transforms;

        ycocg_2_rgb(&r_y, &g_u, &b_v );
        scale_bias_bd8_synthesis< no_scale, bd8 >(&r_y, &g_u, &b_v);

        element->color[0] = r_y;
        element->color[1] = g_u;
        element->color[2] = b_v;
    }

    std::shared_ptr< image > make_rgb( std::shared_ptr<ycocg_image> img )
    {
        auto w              = get_y( *img )->get_width();
        auto h              = get_y( *img)->get_height();

        auto rgb_row_pitch  = (w * 24 + 7) / 8; 
        auto rgb_image_size = rgb_row_pitch * h;

        //auto size           = w * h * sizeof(int32_t);
        
        auto rgb_buffer     = cuda::make_memory_buffer (  rgb_image_size) ;

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        auto a = get_y(img);
        auto b = get_co(img);
        auto c = get_cg(img);

        auto data_a = get_data(a);
        auto data_b = get_data(b);
        auto data_c = get_data(c);

        scale_compose_ycocg_kernel<<<blocks, threads_per_block>>>( reinterpret_cast< uint32_t*> ( data_a )  , reinterpret_cast<uint32_t*> (data_b ), reinterpret_cast<uint32_t*> ( data_c ), *rgb_buffer, w, rgb_row_pitch );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        //return std::make_shared<image> ( image::format_24bpp_rgb, rgb_row_pitch, w, h, std::move(rgb_buffer) ) ;

        return std::shared_ptr<image> ( new image (image::format_24bpp_rgb, rgb_row_pitch, w, h, rgb_buffer ) );
    }

    __global__ void scale_decompose_yuv_kernel( const rgb* in, uint32_t* y_color, uint32_t* u_color, uint32_t* v_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_u  = element->color[1];
        jpegxr::transforms::pixel b_v  = element->color[2];

        using namespace jpegxr::transforms;

        scale_bias_bd8_analysis< no_scale, bd8 >(&r_y, &g_u, &b_v);
        rgb_2_yuv(&r_y, &g_u, &b_v );

        y_color[ row * write_pitch + col ] = r_y;
        u_color[ row * write_pitch + col ] = g_u;
        v_color[ row * write_pitch + col ] = b_v;
    }

    std::shared_ptr<ycbcr_image> make_yuv ( std::shared_ptr<image> image ) 
    {
        auto w         = image->get_width();
        auto h         = image->get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = cuda::make_memory_buffer( size );
        auto u_buffer  = cuda::make_memory_buffer( size );
        auto v_buffer  = cuda::make_memory_buffer( size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_decompose_yuv_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<rgb*> ( image->get() ), *y_buffer, *u_buffer, *v_buffer, image->get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        //debug purposes
        /*
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size  , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), u_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), v_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        */
        return std::make_shared<ycbcr_image> ( make_image_2d (y_buffer, w, h, w),  make_image_2d (u_buffer, w, h, w),  make_image_2d (v_buffer, w, h, w)  );
    }

    __global__ void scale_compose_yuv_kernel( const uint32_t* y_color, const uint32_t* u_color, const uint32_t* v_color, rgb* out, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<rgb*> (  (uint8_t*) out + ( row * write_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y = y_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel g_u = u_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel b_v = v_color[ row * read_pitch + col ];

        using namespace jpegxr::transforms;

        yuv_2_rgb(&r_y, &g_u, &b_v );
        scale_bias_bd8_synthesis< no_scale, bd8 >(&r_y, &g_u, &b_v);

        element->color[0] = r_y;
        element->color[1] = g_u;
        element->color[2] = b_v;
    }

    std::shared_ptr< image > make_rgb( std::shared_ptr<ycbcr_image> img)
    {
        auto w              = get_y(*img)->get_width();
        auto h              = get_y(*img)->get_height();

        auto rgb_row_pitch  = (w * 24 + 7) / 8; 
        auto rgb_image_size = rgb_row_pitch * h;

        auto rgb_buffer     = cuda::make_memory_buffer (  rgb_image_size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_compose_yuv_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<uint32_t*> ( get_data ( *get_y(img) ) ), reinterpret_cast<uint32_t*> ( get_data (*get_cb( img ) ) ), reinterpret_cast<uint32_t*> ( get_data ( *get_cr(img) ) ) , *rgb_buffer, w, rgb_row_pitch );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        //return std::make_shared<image> ( image::format_24bpp_rgb, rgb_row_pitch, w, h, std::move(rgb_buffer) ) ;
        return std::shared_ptr<image> ( new image (image::format_24bpp_rgb, rgb_row_pitch, w, h, std::move(rgb_buffer) ) );
    }

}

static void block_shuffle444(int*data)
{
    int32_t tmp[256];

    int32_t idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[idx+0] = data[ptr+0];
        tmp[idx+1] = data[ptr+1];
        tmp[idx+2] = data[ptr+2];
        tmp[idx+3] = data[ptr+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

static void unblock_shuffle444(int*data)
{
    int tmp[256];

    int idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[ptr+0] = data[idx+0];
        tmp[ptr+1] = data[idx+1];
        tmp[ptr+2] = data[idx+2];
        tmp[ptr+3] = data[idx+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

namespace example
{

}

int32_t main()
{
    try
    {
        auto com_initializer  =  os::windows::com_initializer();
        auto cuda_initializer = example::cuda_initializer();
        auto image  =  example::create_image ( L"test_32x32.png" );
        
        auto yuv  = make_ycocg(image);
        auto back = make_rgb(yuv);

        if ( cuda::is_equal( image->get_buffer(), back->get_buffer() ) )
        {
            std::cout <<"Prefect color transformation" << std::endl;
        }

        auto w      = get_y(yuv)->get_width();
        auto h      = get_y(yuv)->get_height();
        auto pitch  = get_y(yuv)->get_width();

        //
        jpegxr::prefilter2x2_edge( *get_y( yuv ) , w, h, pitch );
        jpegxr::prefilter4x4( *get_y(yuv), w, h, pitch );
        jpegxr::prefilter4_horizontal( *get_y(yuv) , w, h, pitch );
        jpegxr::prefilter4_vertical( *get_y(yuv), w, h, pitch );
        jpegxr::pct4x4( *get_y(yuv), w, h, pitch );

        jpegxr::prefilter2x2_edge( *get_co( yuv ) , w, h, pitch );
        jpegxr::prefilter4x4( *get_co(yuv), w, h, pitch );
        jpegxr::prefilter4_horizontal( *get_co(yuv) , w, h, pitch );
        jpegxr::prefilter4_vertical( *get_co(yuv), w, h, pitch );
        jpegxr::pct4x4( *get_co(yuv), w, h, pitch );

        jpegxr::prefilter2x2_edge( *get_cg( yuv ) , w, h, pitch );
        jpegxr::prefilter4x4( *get_cg(yuv), w, h, pitch );
        jpegxr::prefilter4_horizontal( *get_cg(yuv) , w, h, pitch );
        jpegxr::prefilter4_vertical( *get_cg(yuv), w, h, pitch );
        jpegxr::pct4x4( *get_cg(yuv), w, h, pitch );

        jpegxr::ipct4x4( *get_y(yuv), w, h, pitch );
        jpegxr::overlapfilter4_vertical( *get_y(yuv), w, h, pitch );
        jpegxr::overlapfilter4_horizontal( *get_y(yuv), w, h, pitch );
        jpegxr::overlapfilter4x4( *get_y(yuv), w, h, pitch );
        jpegxr::overlapfilter2x2_edge( *get_y(yuv), w, h, pitch );

        jpegxr::ipct4x4( *get_co(yuv), w, h, pitch );
        jpegxr::overlapfilter4_vertical( *get_co(yuv), w, h, pitch );
        jpegxr::overlapfilter4_horizontal( *get_co(yuv), w, h, pitch );
        jpegxr::overlapfilter4x4( *get_co(yuv), w, h, pitch );
        jpegxr::overlapfilter2x2_edge( *get_co(yuv), w, h, pitch );

        jpegxr::ipct4x4( *get_cg(yuv), w, h, pitch );
        jpegxr::overlapfilter4_vertical( *get_cg(yuv), w, h, pitch );
        jpegxr::overlapfilter4_horizontal( *get_cg(yuv), w, h, pitch );
        jpegxr::overlapfilter4x4( *get_cg(yuv), w, h, pitch );
        jpegxr::overlapfilter2x2_edge( *get_cg(yuv), w, h, pitch );
        

        auto image_out = make_rgb(yuv);

        if ( cuda::is_equal( image->get_buffer(), image_out->get_buffer() ) )
        {
            std::cout <<"Prefect reconstruction." << std::endl;
        }
        else
        {
            std::cout <<"Error in reconstruction." << std::endl;
        }
    }

    catch (const cuda::exception& e)
    {
        std::cerr<<e.what()<<std::endl;
        return 1;
    }

    return 0;
}
