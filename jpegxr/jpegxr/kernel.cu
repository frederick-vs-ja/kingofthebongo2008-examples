#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "cuda_helper.h"

#include <iostream>
#include <memory>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    cuda::throw_if_failed<cuda::exception> ( addWithCuda(c, a, b, arraySize) );

    std::cout << "{1,2,3,4,5} + {10,20,30,40,50} = "<< std::endl << c[0] << c[1] << c[2] << c[3] << c[4];

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cuda::throw_if_failed<cuda::exception> ( hipDeviceReset() );

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    cuda::throw_if_failed<cuda::exception> (  hipSetDevice(0) );

    // Allocate GPU buffers for three vectors (two input, one output)    .
    auto dev_a = std::make_shared< cuda::memory_buffer > ( size * sizeof( int )  );
    auto dev_b = std::make_shared< cuda::memory_buffer > ( size * sizeof( int )  );
    auto dev_c = std::make_shared< cuda::memory_buffer > ( size * sizeof( int )  );

    // Copy input vectors from host memory to GPU buffers.
    cuda::throw_if_failed<cuda::exception> ( hipMemcpy(dev_a->get(), a, size * sizeof(int), hipMemcpyHostToDevice) );
    cuda::throw_if_failed<cuda::exception> ( hipMemcpy(dev_b->get(), b, size * sizeof(int), hipMemcpyHostToDevice) );

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>( *dev_c, *dev_a, *dev_b );

    // Check for any errors launching the kernel
    cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
   
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

    // Copy output vector from GPU buffer to host memory.
    cuda::throw_if_failed<cuda::exception> ( hipMemcpy(c, dev_c->get(), size * sizeof(int), hipMemcpyDeviceToHost) );

    return hipSuccess;
}
