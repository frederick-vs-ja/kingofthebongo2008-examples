#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <exception>
#include <iostream>
#include <memory>

namespace cuda
{
    class exception : public std::exception
    {
        public:

        exception( hipError_t error ) : m_error(error)
        {

        }

        const char * what() const override
        {
            return hipGetErrorString(m_error);
        }

        private:

        hipError_t m_error;
    };

    template < typename exception > void throw_if_failed( hipError_t error )
    {
        if (error != hipSuccess)
        {
            throw exception(error);
        }
    }

    void* malloc( std::size_t size )
    {
        void* r = nullptr;
        auto status = hipMalloc( &r, size );
        if ( status == hipSuccess )
        {
            return r;
        }
        else
        {
            return nullptr;
        }
    }

    inline void* allocate( std::size_t size, void* p )
    {
        auto r = malloc(size);
        if ( r == nullptr )
        {
            throw std::bad_alloc();
        }
        return r;
    }

    template <typename t> inline t* allocate(std::size_t size)
    {
        return reinterpret_cast<t*>(allocate(size, nullptr));
    }

    void  free( void* pointer )
    {
        hipFree( pointer );
    }

    class memory_buffer
    {
        private:

        typedef memory_buffer   this_type;
        int*    m_value;

        void swap(memory_buffer & rhs)
        {
            int* tmp = m_value;
            m_value = rhs.m_value;
            rhs.m_value = tmp;
        }

        public:

        memory_buffer ( int size ) :
        m_value( allocate<int>(size) )
        {

        }

        memory_buffer ( memory_buffer&& rhs ) : m_value(rhs.m_value)
        {
            rhs.m_value = nullptr;
        }

        memory_buffer & operator=(memory_buffer && rhs)
        {
            this_type( static_cast< memory_buffer && >( rhs ) ).swap(*this);
            return *this;
        }


        ~memory_buffer()
        {
            free(m_value);
        }

        const void*    get() const
        {
            return m_value;
        }

        void*    get()
        {
            return m_value;
        }

        template <typename t> operator t*()
        {
            return reinterpret_cast<t*> (m_value);
        }

        template <typename t> operator const t*() const
        {
            return reinterpret_cast<t*> (m_value);
        }

        private:

        memory_buffer( const memory_buffer& );
        memory_buffer& operator=(const memory_buffer&);
    };
}


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    cuda::throw_if_failed<cuda::exception> ( addWithCuda(c, a, b, arraySize) );

    std::cout << "{1,2,3,4,5} + {10,20,30,40,50} = "<< std::endl << c[0] << c[1] << c[2] << c[3] << c[4];

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cuda::throw_if_failed<cuda::exception> ( hipDeviceReset() );

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    cuda::throw_if_failed<cuda::exception> (  hipSetDevice(0) );

    // Allocate GPU buffers for three vectors (two input, one output)    .
    auto dev_a = std::make_shared< cuda::memory_buffer > ( size * sizeof( int )  );
    auto dev_b = std::make_shared< cuda::memory_buffer > ( size * sizeof( int )  );
    auto dev_c = std::make_shared< cuda::memory_buffer > ( size * sizeof( int )  );

    // Copy input vectors from host memory to GPU buffers.
    cuda::throw_if_failed<cuda::exception> ( hipMemcpy(dev_a->get(), a, size * sizeof(int), hipMemcpyHostToDevice) );
    cuda::throw_if_failed<cuda::exception> ( hipMemcpy(dev_b->get(), b, size * sizeof(int), hipMemcpyHostToDevice) );

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>( *dev_c, *dev_a, *dev_b );

    // Check for any errors launching the kernel
    cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
   
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

    // Copy output vector from GPU buffer to host memory.
    cuda::throw_if_failed<cuda::exception> ( hipMemcpy(c, dev_c->get(), size * sizeof(int), hipMemcpyDeviceToHost) );

    return hipSuccess;
}
