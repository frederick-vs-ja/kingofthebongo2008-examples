#include "hip/hip_runtime.h"
﻿#include "precompiled.h"

#include "img_utils.h"

#include <cstdint>
#include <fstream>
#include <iostream>
#include <memory>

#include <util/util_memory.h>

#include <hip/hip_runtime.h>
#include <>
#include <device_functions_decls.h>
#include <hip/hip_vector_types.h>

#include <jxr/cuda_helper.h>
#include <jxr/jxr_transforms.h>
#include <jxr/jxr_analysis.h>

#include <jxr/jxr_filter.h>
#include <jxr/jxr_prefilter.h>
#include <jxr/jxr_overlapfilter.h>
#include <jxr/jxr_pct.h>
#include <jxr/jxr_decompose.h>

#include <os/windows/com_initializer.h>

#include "img_images.h"
#include "img_loader.h"

namespace example
{
    class cuda_initializer
    {
        public:
        cuda_initializer()
        {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cuda::throw_if_failed<cuda::exception> (  hipSetDevice(0) );
        }

        ~cuda_initializer()
        {
            // hipDeviceReset must be called before exiting in order for profiling and
            // tracing tools such as Nsight and Visual Profiler to show complete traces.
            cuda::throw_if_failed<cuda::exception> ( hipDeviceReset() );
        }
    };
}

namespace example
{
    struct rgb 
    {
        uint8_t color[3];
    };

    __global__ void scale_decompose_ycocg_kernel( const rgb* in, jpegxr::transforms::pixel* y_color, jpegxr::transforms::pixel* co_color, jpegxr::transforms::pixel* cg_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_co = element->color[1];
        jpegxr::transforms::pixel b_cg = element->color[2];

        using namespace jpegxr::transforms;

        scale_bias_bd8_analysis< no_scale, bd8 >(&r_y, &g_co, &b_cg);
        rgb_2_ycocg(&r_y, &g_co, &b_cg );

        y_color [ row * write_pitch + col ] = r_y;
        co_color[ row * write_pitch + col ] = g_co;
        cg_color[ row * write_pitch + col ] = b_cg;
    }

    std::shared_ptr< ycocg_image > make_ycocg ( std::shared_ptr<image> image ) 
    {
        auto w         = image->get_width();
        auto h         = image->get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = cuda::make_memory_buffer ( size );
        auto co_buffer = cuda::make_memory_buffer ( size );
        auto cg_buffer = cuda::make_memory_buffer ( size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_decompose_ycocg_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<rgb*> ( image->get() ), *y_buffer, *co_buffer, *cg_buffer, image->get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );


        /*
        //debug purposes
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), co_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), cg_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        */

        return std::make_shared<ycocg_image> ( make_image_2d ( y_buffer, w, h, w ), make_image_2d (co_buffer, w, h, w) , make_image_2d( cg_buffer, w, h, w ) ) ;
    }

    __global__ void scale_compose_ycocg_kernel( const jpegxr::transforms::pixel* y_color, const jpegxr::transforms::pixel* u_color, const jpegxr::transforms::pixel* v_color, rgb* out, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<rgb*> (  (uint8_t*) out + ( row * write_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y = y_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel g_u = u_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel b_v = v_color[ row * read_pitch + col ];

        using namespace jpegxr::transforms;

        ycocg_2_rgb(&r_y, &g_u, &b_v );
        scale_bias_bd8_synthesis< no_scale, bd8 >(&r_y, &g_u, &b_v);

        r_y = r_y < 0 ? 0 : r_y;
        g_u = g_u < 0 ? 0 : g_u;
        b_v = b_v < 0 ? 0 : b_v;

        r_y = r_y > 255 ? 255 : r_y;
        g_u = g_u > 255 ? 255 : g_u;
        b_v = b_v > 255 ? 255 : b_v;

        element->color[0] = r_y;
        element->color[1] = g_u;
        element->color[2] = b_v;
    }

    std::shared_ptr< image > make_rgb( std::shared_ptr<ycocg_image> img )
    {
        auto w              = get_y( *img )->get_width();
        auto h              = get_y( *img)->get_height();

        auto rgb_row_pitch  = (w * 24 + 7) / 8; 
        auto rgb_image_size = rgb_row_pitch * h;

       
        auto rgb_buffer     = cuda::make_memory_buffer (  rgb_image_size) ;

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_compose_ycocg_kernel<<<blocks, threads_per_block>>>( get_pixels( get_y( img ) ), get_pixels( get_co(img) ), get_pixels( get_cg(img) ), *rgb_buffer, w, rgb_row_pitch );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        return std::shared_ptr<image> ( new image (image::format_24bpp_rgb, rgb_row_pitch, w, h, std::move(rgb_buffer) ) );
    }

    std::shared_ptr<image_2d> make_low_pass( std::shared_ptr<image_2d> img )
    {
        auto w              = img -> get_width() / 4;
        auto h              = img -> get_height() / 4;
        auto pitch          = w;
        auto image_size     = pitch * h * sizeof(jpegxr::transforms::pixel);

        auto buffer         = cuda::make_memory_buffer ( image_size ) ;

        auto kernel_params      = cuda::make_threads_blocks_16( w, h );

        jpegxr::decompose::split_lp_hp <<< std::get<0>( kernel_params), std::get<1>(kernel_params) >>> ( get_pixels( img ), *buffer,  img->get_pitch(), w, h, pitch ); 

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        return make_image_2d( buffer, w, h, pitch) ;
    }

    __global__ void scale_decompose_yuv_kernel( const rgb* in, uint32_t* y_color, uint32_t* u_color, uint32_t* v_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_u  = element->color[1];
        jpegxr::transforms::pixel b_v  = element->color[2];

        using namespace jpegxr::transforms;

        scale_bias_bd8_analysis< no_scale, bd8 >(&r_y, &g_u, &b_v);
        rgb_2_yuv(&r_y, &g_u, &b_v );

        y_color[ row * write_pitch + col ] = r_y;
        u_color[ row * write_pitch + col ] = g_u;
        v_color[ row * write_pitch + col ] = b_v;
    }

    std::shared_ptr<ycbcr_image> make_yuv ( std::shared_ptr<image> image ) 
    {
        auto w         = image->get_width();
        auto h         = image->get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = cuda::make_memory_buffer( size );
        auto u_buffer  = cuda::make_memory_buffer( size );
        auto v_buffer  = cuda::make_memory_buffer( size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_decompose_yuv_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<rgb*> ( image->get() ), *y_buffer, *u_buffer, *v_buffer, image->get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        //debug purposes
        /*
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size  , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), u_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), v_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        */
        return std::make_shared<ycbcr_image> ( make_image_2d (y_buffer, w, h, w),  make_image_2d (u_buffer, w, h, w),  make_image_2d (v_buffer, w, h, w)  );
    }

    __global__ void scale_compose_yuv_kernel( const uint32_t* y_color, const uint32_t* u_color, const uint32_t* v_color, rgb* out, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<rgb*> (  (uint8_t*) out + ( row * write_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y = y_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel g_u = u_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel b_v = v_color[ row * read_pitch + col ];

        using namespace jpegxr::transforms;

        yuv_2_rgb(&r_y, &g_u, &b_v );
        scale_bias_bd8_synthesis< no_scale, bd8 >(&r_y, &g_u, &b_v);

        r_y = r_y < 0 ? 0 : r_y;
        g_u = g_u < 0 ? 0 : g_u;
        b_v = b_v < 0 ? 0 : b_v;

        r_y = r_y > 255 ? 255 : r_y;
        g_u = g_u > 255 ? 255 : g_u;
        b_v = b_v > 255 ? 255 : b_v;

        element->color[0] = r_y;
        element->color[1] = g_u;
        element->color[2] = b_v;
    }

    std::shared_ptr< image > make_rgb( std::shared_ptr<ycbcr_image> img)
    {
        auto w              = get_y(*img)->get_width();
        auto h              = get_y(*img)->get_height();

        auto rgb_row_pitch  = (w * 24 + 7) / 8; 
        auto rgb_image_size = rgb_row_pitch * h;

        auto rgb_buffer     = cuda::make_memory_buffer (  rgb_image_size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_compose_yuv_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<uint32_t*> ( get_data ( *get_y(img) ) ), reinterpret_cast<uint32_t*> ( get_data (*get_cb( img ) ) ), reinterpret_cast<uint32_t*> ( get_data ( *get_cr(img) ) ) , *rgb_buffer, w, rgb_row_pitch );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        return std::make_shared<image> ( image::format_24bpp_rgb, rgb_row_pitch, w, h, std::move(rgb_buffer) ) ;
    }
}

static void block_shuffle444(int*data)
{
    int32_t tmp[256];

    int32_t idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[idx+0] = data[ptr+0];
        tmp[idx+1] = data[ptr+1];
        tmp[idx+2] = data[ptr+2];
        tmp[idx+3] = data[ptr+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

static void unblock_shuffle444(int*data)
{
    int tmp[256];

    int idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[ptr+0] = data[idx+0];
        tmp[ptr+1] = data[idx+1];
        tmp[ptr+2] = data[idx+2];
        tmp[ptr+3] = data[idx+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

namespace example
{

}

int32_t main()
{
    try
    {
        auto com_initializer  =  os::windows::com_initializer();
        auto cuda_initializer = example::cuda_initializer();
        auto image  =  example::create_image ( L"test_32x32.png" );

        auto image1  =  example::make_test_image_linear_16x16();

        
        print_image(image1);

        return 0;
                
        auto yuv  = make_ycocg(image);

        auto y      = get_y(yuv);
        auto w      = y->get_width();
        auto h      = y->get_height();
        auto pitch  = y->get_width();

        std::cout<<"YCoCg...."<<std::endl;
        print_image ( y  );
        //
        jpegxr::prefilter2x2_edge( *y , w, h, pitch );
        jpegxr::prefilter4x4( *y, w, h, pitch );
        jpegxr::prefilter4_horizontal( *y , w, h, pitch );
        jpegxr::prefilter4_vertical( *y, w, h, pitch );
        jpegxr::pct4x4( *y, w, h, pitch );

        std::cout<<"Stage 1..."<<std::endl;
        print_image ( y  );

        auto lp = make_low_pass( y ) ;

        std::cout<<"Low pass..."<<std::endl;
        print_image ( lp  );

        jpegxr::pct4x4( *lp, lp->get_width(), lp->get_height(), lp->get_pitch() );
        std::cout<<"Second Stage..."<<std::endl;

        print_image ( lp  );

        //auto lp = make_low_pass(yuv);

        jpegxr::ipct4x4( *y, w, h, pitch );
        jpegxr::overlapfilter4_vertical( *y, w, h, pitch );
        jpegxr::overlapfilter4_horizontal( *y, w, h, pitch );
        jpegxr::overlapfilter4x4( *y, w, h, pitch );
        jpegxr::overlapfilter2x2_edge( *y, w, h, pitch );

        auto image_out = make_rgb(yuv);

        //print_image ( example::make_test_image_2x2( 16, 16, 0, 1, 2, 3) );

        if ( cuda::is_equal( image->get_buffer(), image_out->get_buffer() ) )
        {
            std::cout <<"Prefect reconstruction." << std::endl;
        }
        else
        {
            std::cout <<"Error in reconstruction." << std::endl;
        }
    }

    catch (const cuda::exception& e)
    {
        std::cerr<<e.what()<<std::endl;
        return 1;
    }

    return 0;
}
