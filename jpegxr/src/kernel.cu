#include "hip/hip_runtime.h"
﻿#include <cstdint>
#include <fstream>
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>

#include <jxr/cuda_helper.h>
#include <jxr/jxr_transforms.h>
#include <jxr/jxr_analysis.h>

#include <jxr/jxr_filter.h>
#include <jxr/jxr_prefilter.h>
#include <jxr/jxr_overlapfilter.h>
#include <jxr/jxr_pct.h>
#include <jxr/jxr_decompose.h>

#include <os/windows/com_initializer.h>

#include "img_images.h"
#include "img_loader.h"

namespace example
{
    class cuda_initializer
    {
        public:
        cuda_initializer()
        {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cuda::throw_if_failed<cuda::exception> (  hipSetDevice(0) );
        }

        ~cuda_initializer()
        {
            // hipDeviceReset must be called before exiting in order for profiling and
            // tracing tools such as Nsight and Visual Profiler to show complete traces.
            cuda::throw_if_failed<cuda::exception> ( hipDeviceReset() );
        }
    };
}

namespace example
{
    struct rgb 
    {
        uint8_t color[3];
    };

    __global__ void decompose_ycocg_kernel( const rgb* in, uint32_t* y_color, uint32_t* co_color, uint32_t* cg_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_co = element->color[1];
        jpegxr::transforms::pixel b_cg = element->color[2];

        using namespace jpegxr::transforms;

        rgb_2_ycocg(&r_y, &g_co, &b_cg );

        y_color [ row * write_pitch + col ] = r_y;
        co_color[ row * write_pitch + col ] = g_co;
        cg_color[ row * write_pitch + col ] = b_cg;
    }

    
    ycocg_image decompose_ycocg ( const image& image ) 
    {
        auto w         = image.get_width();
        auto h         = image.get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = std::make_shared < cuda::memory_buffer > ( cuda::allocate<void*> ( size ) );
        auto co_buffer = std::make_shared < cuda::memory_buffer > ( cuda::allocate<void*> ( size ) );
        auto cg_buffer = std::make_shared < cuda::memory_buffer > ( cuda::allocate<void*> ( size ) );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        decompose_ycocg_kernel<<<blocks, threads_per_block>>>( image, *y_buffer, *co_buffer, *cg_buffer, image.get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        //debug purposes
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), co_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), cg_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto row = 15;
        auto col = 15;
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        auto el1 = res1[ row * w + col ];

        return ycocg_image ( y_buffer, co_buffer, cg_buffer, w, h, w );
    }


    __global__ void decompose_yuv_kernel( const rgb* in, uint32_t* y_color, uint32_t* u_color, uint32_t* v_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_u  = element->color[1];
        jpegxr::transforms::pixel b_v  = element->color[2];

        using namespace jpegxr::transforms;

        rgb_2_yuv(&r_y, &g_u, &b_v );

        y_color[ row * write_pitch + col ] = r_y;
        u_color[ row * write_pitch + col ] = g_u;
        v_color[ row * write_pitch + col ] = b_v;
    }

    
    ycbcr_image decompose_yuv ( const image& image ) 
    {
        auto w         = image.get_width();
        auto h         = image.get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = std::make_shared < cuda::memory_buffer > ( cuda::allocate<void*> ( size ) );
        auto u_buffer = std::make_shared < cuda::memory_buffer > ( cuda::allocate<void*> ( size ) );
        auto v_buffer = std::make_shared < cuda::memory_buffer > ( cuda::allocate<void*> ( size ) );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        decompose_ycocg_kernel<<<blocks, threads_per_block>>>( image, *y_buffer, *u_buffer, *v_buffer, image.get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        //debug purposes
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), u_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), v_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto row = 15;
        auto col = 15;
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        auto el1 = res1[ row * w + col ];

        return ycbcr_image ( y_buffer, u_buffer, v_buffer, w, h, w );
    }
}

static void block_shuffle444(int*data)
{
    int32_t tmp[256];

    int32_t idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[idx+0] = data[ptr+0];
        tmp[idx+1] = data[ptr+1];
        tmp[idx+2] = data[ptr+2];
        tmp[idx+3] = data[ptr+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

static void unblock_shuffle444(int*data)
{
    int tmp[256];

    int idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[ptr+0] = data[idx+0];
        tmp[ptr+1] = data[idx+1];
        tmp[ptr+2] = data[idx+2];
        tmp[ptr+3] = data[idx+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

int32_t main()
{
    try
    {
        auto com_initializer  =  os::windows::com_initializer();
        auto cuda_initializer = example::cuda_initializer();
        auto image  =  example::create_image ( L"test_32x32.png" );

        int32_t data[256];

        for (int32_t i = 0; i < 256; ++i )
        {
            data[i] = i;
        }

        block_shuffle444(data);
        unblock_shuffle444(data);

        for (int32_t i = 0; i <255; ++i)
        {
            for (int32_t j = 0; j < 255; ++j)
            {
                for (int32_t k = 0; k < 255; ++k )
                {
                    auto r0 = i;
                    auto g0 = j;
                    auto b0 = k;

                    auto r1 = i;
                    auto g1 = j;
                    auto b1 = k;

                    using namespace jpegxr::transforms;

                    rgb_2_yuv(&r0, &g0, &b0 );
                    rgb_2_ycocg(&r1, &g1, &b1 );

                    if (! (  r0 == r1 && g0==g1 && b0==b1) )
                    {
                        //__debugbreak();
                    }
                }
            }
        }


        auto yuv = decompose_ycocg(*image);

        auto w      = yuv.get_width();
        auto h      = yuv.get_height();
        auto pitch  = yuv.get_width();
        auto size   = w * h * sizeof(jpegxr::transforms::pixel) ;

        auto copy_of_y_1  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( copy_of_y_1.get(),  *yuv.get_y(), size   , hipMemcpyDeviceToHost) );

        jpegxr::prefilter2x2_edge( yuv.get_y(), w, h, pitch );
        jpegxr::prefilter4x4( yuv.get_y(), w, h, pitch );
        jpegxr::prefilter4_horizontal( yuv.get_y(), w, h, pitch );
        jpegxr::prefilter4_vertical( yuv.get_y(), w, h, pitch );
        jpegxr::pct4x4( yuv.get_y(), w, h, pitch );

        jpegxr::ipct4x4( yuv.get_y(), w, h, pitch );
        jpegxr::overlapfilter2x2_edge( yuv.get_y(), w, h, pitch );
        jpegxr::overlapfilter4x4( yuv.get_y(), w, h, pitch );
        jpegxr::overlapfilter4_horizontal( yuv.get_y(), w, h, pitch );
        jpegxr::overlapfilter4_vertical( yuv.get_y(), w, h, pitch );

        auto copy_of_y_2  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( copy_of_y_2.get(),  *yuv.get_y(), size   , hipMemcpyDeviceToHost) );

        auto result = std::memcmp ( copy_of_y_1.get(), copy_of_y_2.get(), size );

        if (result == 0 )
        {
            std::cout <<"Prefect reconstruction." << std::endl;
        }
        else
        {
            std::cerr <<"Error in reconstruction." << std::endl;
        }
    }

    catch (const cuda::exception& e)
    {
        std::cerr<<e.what()<<std::endl;
        return 1;
    }

    return 0;
}
