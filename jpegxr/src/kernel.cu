#include "hip/hip_runtime.h"
﻿#include "precompiled.h"

#include <cstdint>
#include <fstream>
#include <iostream>
#include <memory>

#include <util/util_memory.h>

#include <hip/hip_runtime.h>
#include <>
#include <device_functions_decls.h>
#include <hip/hip_vector_types.h>

#include <jxr/cuda_helper.h>
#include <jxr/jxr_transforms.h>
#include <jxr/jxr_analysis.h>

#include <jxr/jxr_filter.h>
#include <jxr/jxr_prefilter.h>
#include <jxr/jxr_overlapfilter.h>
#include <jxr/jxr_pct.h>
#include <jxr/jxr_decompose.h>

#include <os/windows/com_initializer.h>

#include "img_images.h"
#include "img_loader.h"

namespace example
{
    class cuda_initializer
    {
        public:
        cuda_initializer()
        {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cuda::throw_if_failed<cuda::exception> (  hipSetDevice(0) );
        }

        ~cuda_initializer()
        {
            // hipDeviceReset must be called before exiting in order for profiling and
            // tracing tools such as Nsight and Visual Profiler to show complete traces.
            cuda::throw_if_failed<cuda::exception> ( hipDeviceReset() );
        }
    };

    const jpegxr::transforms::pixel* get_pixels( const image_2d& image )
    {
        return reinterpret_cast<const jpegxr::transforms::pixel*> ( get_data(image) );
    }

    jpegxr::transforms::pixel* get_pixels( image_2d& image )
    {
        return reinterpret_cast<jpegxr::transforms::pixel*> ( get_data(image) );
    }

    jpegxr::transforms::pixel* get_pixels( const std::shared_ptr<image_2d> image )
    {
        return reinterpret_cast<jpegxr::transforms::pixel*> ( get_data(image) );
    }

    __global__ void make_test_image_kernel( jpegxr::transforms::pixel* pixels, const uint32_t pixel_value, const uint32_t width, const uint32_t height, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;

        if ( row < 0  && row > ( height - 1) )
        {
            return;
        }

        if ( col < 0 || col > ( width - 1 ) ) 
        {
            return;
        }

        pixels [ row * write_pitch + col ] = pixel_value;
    }

    __global__ void make_test_image_kernel_2x2( jpegxr::transforms::pixel* pixels, const uint32_t lt, const uint32_t rt, const uint32_t lb, const uint32_t rb, const uint32_t width, const uint32_t height, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = 2 * y;
        auto col = 2 * x;

        if ( row < 0  && row > ( height - 1) )
        {
            return;
        }

        if ( col < 0 || col > ( width - 1 ) ) 
        {
            return;
        }

        pixels [ row * write_pitch + col ] = lt;
        pixels [ row * write_pitch + col + 1] = rt;

        pixels [ (row + 1) * write_pitch + col ] = lb;
        pixels [ (row + 1) * write_pitch + col + 1 ] = rb;

        
    }

    std::pair< dim3, dim3> make_threads_blocks_16 ( uint32_t pixel_width, uint32_t pixel_height )
    {
        auto w = pixel_width;
        auto h = pixel_height;

        return std::make_pair( dim3 ( w, h,  1 ), dim3 ( ( w + 15 )  / 16 , ( h + 15 ) / 16, 1 ) );
    }

    std::shared_ptr< image_2d > make_test_image( uint32_t width, uint32_t height, jpegxr::transforms::pixel pixel_value)
    {
        auto image_size = width * height * sizeof(jpegxr::transforms::pixel);

        auto w                  = width;
        auto h                  = height;
        auto pitch              = w;

        auto kernel_params      = make_threads_blocks_16( w, h );
        

        auto buffer             = cuda::make_memory_buffer (  image_size );

        make_test_image_kernel<<< std::get<0>( kernel_params), std::get<1>(kernel_params) >>> ( *buffer, pixel_value, w, h, pitch );

        return make_image_2d( buffer, width, height, width );
    }

    std::shared_ptr< image_2d > make_test_image_2x2( uint32_t width, uint32_t height, jpegxr::transforms::pixel lt, jpegxr::transforms::pixel rt, jpegxr::transforms::pixel lb, jpegxr::transforms::pixel rb)
    {
        auto image_size = width * height * sizeof(jpegxr::transforms::pixel);

        auto w                  = width;
        auto h                  = height;
        auto pitch              = w;

        auto kernel_params      = make_threads_blocks_16( w, h );
        

        auto buffer             = cuda::make_memory_buffer (  image_size );

        make_test_image_kernel_2x2<<< std::get<0>( kernel_params), std::get<1>(kernel_params) >>> ( *buffer, lt, rt, lb, rb, w, h, pitch );

        return make_image_2d( buffer, width, height, width );
    }

    std::shared_ptr< image_2d > make_zero_image( uint32_t width, uint32_t height, jpegxr::transforms::pixel pixel_value)
    {
        return make_test_image( width, height, 0 );
    }

    void print_image( std::shared_ptr<image_2d> image )
    {
        auto size               = image->get_width() * image->get_height() * sizeof(jpegxr::transforms::pixel) ;
        auto y                  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        ::cuda::throw_if_failed<::cuda::exception> ( hipMemcpy( y.get(), get_pixels(image), size , hipMemcpyDeviceToHost) );

        auto ptr = reinterpret_cast<jpegxr::transforms::pixel*> (&y[0]);

        for( uint32_t i = 0; i < image->get_height(); ++i )
        {
            for (uint32_t j = 0; j < image->get_width(); ++j)
            {
                std::cout << *( ptr++ ) <<"\t";

                if ( j == image->get_width() - 1 )
                {
                    std::cout<<std::endl;
                }
            }
        }
    }
}

namespace example
{
    struct rgb 
    {
        uint8_t color[3];
    };

    __global__ void scale_decompose_ycocg_kernel( const rgb* in, jpegxr::transforms::pixel* y_color, jpegxr::transforms::pixel* co_color, jpegxr::transforms::pixel* cg_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_co = element->color[1];
        jpegxr::transforms::pixel b_cg = element->color[2];

        using namespace jpegxr::transforms;

        scale_bias_bd8_analysis< no_scale, bd8 >(&r_y, &g_co, &b_cg);
        rgb_2_ycocg(&r_y, &g_co, &b_cg );

        y_color [ row * write_pitch + col ] = r_y;
        co_color[ row * write_pitch + col ] = g_co;
        cg_color[ row * write_pitch + col ] = b_cg;
    }

    std::shared_ptr< ycocg_image > make_ycocg ( std::shared_ptr<image> image ) 
    {
        auto w         = image->get_width();
        auto h         = image->get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = cuda::make_memory_buffer ( size );
        auto co_buffer = cuda::make_memory_buffer ( size );
        auto cg_buffer = cuda::make_memory_buffer ( size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_decompose_ycocg_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<rgb*> ( image->get() ), *y_buffer, *co_buffer, *cg_buffer, image->get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );


        /*
        //debug purposes
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), co_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), cg_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        */

        return std::make_shared<ycocg_image> ( make_image_2d ( y_buffer, w, h, w ), make_image_2d (co_buffer, w, h, w) , make_image_2d( cg_buffer, w, h, w ) ) ;
    }

    __global__ void scale_compose_ycocg_kernel( const jpegxr::transforms::pixel* y_color, const jpegxr::transforms::pixel* u_color, const jpegxr::transforms::pixel* v_color, rgb* out, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<rgb*> (  (uint8_t*) out + ( row * write_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y = y_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel g_u = u_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel b_v = v_color[ row * read_pitch + col ];

        using namespace jpegxr::transforms;

        ycocg_2_rgb(&r_y, &g_u, &b_v );
        scale_bias_bd8_synthesis< no_scale, bd8 >(&r_y, &g_u, &b_v);

        element->color[0] = r_y;
        element->color[1] = g_u;
        element->color[2] = b_v;
    }

    std::shared_ptr< image > make_rgb( std::shared_ptr<ycocg_image> img )
    {
        auto w              = get_y( *img )->get_width();
        auto h              = get_y( *img)->get_height();

        auto rgb_row_pitch  = (w * 24 + 7) / 8; 
        auto rgb_image_size = rgb_row_pitch * h;

        //auto size           = w * h * sizeof(int32_t);
        
        auto rgb_buffer     = cuda::make_memory_buffer (  rgb_image_size) ;

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_compose_ycocg_kernel<<<blocks, threads_per_block>>>( get_pixels( get_y( img ) ), get_pixels( get_co(img) ), get_pixels( get_cg(img) ), *rgb_buffer, w, rgb_row_pitch );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        return std::shared_ptr<image> ( new image (image::format_24bpp_rgb, rgb_row_pitch, w, h, std::move(rgb_buffer) ) );
    }

    std::shared_ptr<image_2d> make_low_pass( std::shared_ptr<image_2d> img )
    {
        auto w              = img -> get_width() / 4;
        auto h              = img -> get_width() / 4;
        auto pitch          = w;
        auto image_size     = pitch * h * sizeof(jpegxr::transforms::pixel);

        auto buffer         = cuda::make_memory_buffer ( image_size ) ;

        auto blocks             = dim3 ( ( w + 15 )  / 16 , ( h + 15 ) / 16, 1 );
        auto threads_per_block  = dim3 ( 16,  16,  1 );

        jpegxr::decompose::split_lp_hp <<< blocks, threads_per_block >>> ( get_pixels( img ), *buffer,  img->get_pitch(), w, h, pitch ); 

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        return make_image_2d( buffer, w, h, pitch) ;
    }

    __global__ void scale_decompose_yuv_kernel( const rgb* in, uint32_t* y_color, uint32_t* u_color, uint32_t* v_color, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<const rgb*> (  (uint8_t*) in + ( row * read_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y  = element->color[0];
        jpegxr::transforms::pixel g_u  = element->color[1];
        jpegxr::transforms::pixel b_v  = element->color[2];

        using namespace jpegxr::transforms;

        scale_bias_bd8_analysis< no_scale, bd8 >(&r_y, &g_u, &b_v);
        rgb_2_yuv(&r_y, &g_u, &b_v );

        y_color[ row * write_pitch + col ] = r_y;
        u_color[ row * write_pitch + col ] = g_u;
        v_color[ row * write_pitch + col ] = b_v;
    }

    std::shared_ptr<ycbcr_image> make_yuv ( std::shared_ptr<image> image ) 
    {
        auto w         = image->get_width();
        auto h         = image->get_height();
        auto size      = w * h * sizeof(int32_t) ;
        
        auto y_buffer  = cuda::make_memory_buffer( size );
        auto u_buffer  = cuda::make_memory_buffer( size );
        auto v_buffer  = cuda::make_memory_buffer( size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_decompose_yuv_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<rgb*> ( image->get() ), *y_buffer, *u_buffer, *v_buffer, image->get_pitch(), w );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        //debug purposes
        /*
        auto y  = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto co = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );
        auto cg = std::unique_ptr< uint8_t[] > ( new uint8_t [ size ] );

        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( y.get(),  y_buffer->get(),  size  , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( co.get(), u_buffer->get(), size   , hipMemcpyDeviceToHost) );
        cuda::throw_if_failed<cuda::exception> ( hipMemcpy( cg.get(), v_buffer->get(), size   , hipMemcpyDeviceToHost) );

        // element access into this image looks like this
        auto res1 = reinterpret_cast<int32_t*> ( y.get()  );
        auto res2 = reinterpret_cast<int32_t*> ( co.get() );
        auto res3 = reinterpret_cast<int32_t*> ( cg.get() );
        */
        return std::make_shared<ycbcr_image> ( make_image_2d (y_buffer, w, h, w),  make_image_2d (u_buffer, w, h, w),  make_image_2d (v_buffer, w, h, w)  );
    }

    __global__ void scale_compose_yuv_kernel( const uint32_t* y_color, const uint32_t* u_color, const uint32_t* v_color, rgb* out, const uint32_t read_pitch, const uint32_t write_pitch )
    {
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        auto row = y;
        auto col = x;
        
        auto element = reinterpret_cast<rgb*> (  (uint8_t*) out + ( row * write_pitch )  + sizeof(rgb) * col ); 

        jpegxr::transforms::pixel r_y = y_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel g_u = u_color[ row * read_pitch + col ];
        jpegxr::transforms::pixel b_v = v_color[ row * read_pitch + col ];

        using namespace jpegxr::transforms;

        yuv_2_rgb(&r_y, &g_u, &b_v );
        scale_bias_bd8_synthesis< no_scale, bd8 >(&r_y, &g_u, &b_v);

        element->color[0] = r_y;
        element->color[1] = g_u;
        element->color[2] = b_v;
    }

    std::shared_ptr< image > make_rgb( std::shared_ptr<ycbcr_image> img)
    {
        auto w              = get_y(*img)->get_width();
        auto h              = get_y(*img)->get_height();

        auto rgb_row_pitch  = (w * 24 + 7) / 8; 
        auto rgb_image_size = rgb_row_pitch * h;

        auto rgb_buffer     = cuda::make_memory_buffer (  rgb_image_size );

        auto blocks = 1;
        auto threads_per_block = dim3( w, h );

        scale_compose_yuv_kernel<<<blocks, threads_per_block>>>( reinterpret_cast<uint32_t*> ( get_data ( *get_y(img) ) ), reinterpret_cast<uint32_t*> ( get_data (*get_cb( img ) ) ), reinterpret_cast<uint32_t*> ( get_data ( *get_cr(img) ) ) , *rgb_buffer, w, rgb_row_pitch );

        cuda::throw_if_failed<cuda::exception> ( hipGetLastError() );
        cuda::throw_if_failed<cuda::exception> ( hipDeviceSynchronize() );

        return std::make_shared<image> ( image::format_24bpp_rgb, rgb_row_pitch, w, h, std::move(rgb_buffer) ) ;
    }
}

static void block_shuffle444(int*data)
{
    int32_t tmp[256];

    int32_t idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[idx+0] = data[ptr+0];
        tmp[idx+1] = data[ptr+1];
        tmp[idx+2] = data[ptr+2];
        tmp[idx+3] = data[ptr+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

static void unblock_shuffle444(int*data)
{
    int tmp[256];

    int idx;
    for (idx = 0 ; idx < 256 ; idx += 4) {
        int blk = idx/16;
        int mbx = blk%4;
        int mby = blk/4;
        int pix = idx%16;
        int py = pix/4;

        int ptr = 16*4*mby + 4*mbx + 16*py;
        tmp[ptr+0] = data[idx+0];
        tmp[ptr+1] = data[idx+1];
        tmp[ptr+2] = data[idx+2];
        tmp[ptr+3] = data[idx+3];
    }

    for (idx = 0 ; idx < 256 ; idx += 1)
        data[idx] = tmp[idx];
}

namespace example
{

}

int32_t main()
{
    try
    {
        auto com_initializer  =  os::windows::com_initializer();
        auto cuda_initializer = example::cuda_initializer();
        auto image  =  example::create_image ( L"test_32x32.png" );
        
        auto yuv  = make_ycocg(image);
        auto back = make_rgb(yuv);

        if ( cuda::is_equal( image->get_buffer(), back->get_buffer() ) )
        {
            std::cout <<"Prefect color transformation" << std::endl;
        }

        auto y      = get_y(yuv);
        auto w      = y->get_width();
        auto h      = y->get_height();
        auto pitch  = y->get_width();

        //
        jpegxr::prefilter2x2_edge( *y , w, h, pitch );
        jpegxr::prefilter4x4( *y, w, h, pitch );
        jpegxr::prefilter4_horizontal( *y , w, h, pitch );
        jpegxr::prefilter4_vertical( *y, w, h, pitch );
        jpegxr::pct4x4( *y, w, h, pitch );

        auto lp = make_low_pass( example::make_test_image( 16, 16, 5) ) ;

        //auto lp = make_low_pass(yuv);

        jpegxr::ipct4x4( *y, w, h, pitch );
        jpegxr::overlapfilter4_vertical( *y, w, h, pitch );
        jpegxr::overlapfilter4_horizontal( *y, w, h, pitch );
        jpegxr::overlapfilter4x4( *y, w, h, pitch );
        jpegxr::overlapfilter2x2_edge( *y, w, h, pitch );

        auto image_out = make_rgb(yuv);

        print_image ( example::make_test_image_2x2( 16, 16, 0, 1, 2, 3) );

        if ( cuda::is_equal( image->get_buffer(), image_out->get_buffer() ) )
        {
            std::cout <<"Prefect reconstruction." << std::endl;
        }
        else
        {
            std::cout <<"Error in reconstruction." << std::endl;
        }
    }

    catch (const cuda::exception& e)
    {
        std::cerr<<e.what()<<std::endl;
        return 1;
    }

    return 0;
}
